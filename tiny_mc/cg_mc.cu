#include "hip/hip_runtime.h"
*\ Calling Functions *\
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "params.h"
#include "photon.cuh"
#include "xorshift32.cuh"

#define PHOTON_CAP 1 << 16
#define THREADS_PER_FRAME 1UL
#define WARPS THREADS_PER_BLOCK/32
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA ERROR %s:%d: %s\n", file, line, hipGetErrorString(code));
        exit(code);
    }
}
#define GL_CHECK() { glCheck(__FILE__, __LINE__); }
inline void glCheck(const char* file, int line)
{
    GLenum err;
    while ((err = glGetError()) != GL_NO_ERROR) {
        fprintf(stderr, "OpenGL ERROR %s:%d: 0x%X\n", file, line, err);
        exit(1);
    }
}


static const char* VSHADER = R"(
#version 430
vec2 vertices[4] = vec2[](
    vec2(-1.0,  1.0),
    vec2(-1.0, -1.0),
    vec2( 1.0, -1.0),
    vec2( 1.0,  1.0)
);
uint indices[6] = uint[](0, 1, 2, 0, 2, 3);
void main() {
    gl_Position = vec4(vertices[indices[gl_VertexID]], 0.0, 1.0);
})";

static const char* FSHADER = R"(
#version 430
#define MC 0.7071067811865476f
out vec4 frag_color;
layout(std430, binding = 0) readonly buffer ssbo {
    float heats[];
} shells;
void main() {
    vec2 uv = gl_FragCoord.xy / vec2(800);
    float dr = length(uv - vec2(0.5));
    int heat_id = int((dr / MC) * float(shells.heats.length() - 1));
    float heat = shells.heats[heat_id];
    float L = 2.0, b = 1.0, k = 0.004;
    float heat_fit = L / (1.0 + b * exp(-k * heat)) - 1.0;
    frag_color = vec4(heat_fit, 0.0, 0.0, 1.0);
})";


__global__ void simulate_kernel(float* __restrict__ heats, float* __restrict__ heats_squared, unsigned long threads_this_frame)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= threads_this_frame) return;
    unsigned int btid = threadIdx.x;
    unsigned int wtid = btid / warpSize;
    //unsigned int lane = btid & (warpSize - 1);

    // Fase 1: Incialización
    __shared__ float heats_local[WARPS][SHELLS];
    __shared__ float heats_squared_local[WARPS][SHELLS];
    Xorshift32 rng;
    xorshift32_init(&rng, SEED ^ (btid * 0x9E3779B9u) ^ (blockIdx.x * 0x85EBCA6Bu) ^ (blockDim.x * 0xC2B2AE35u));
    if (btid == 0) {    
        for (int i = 0; i < WARPS; ++i) {
	    for (int j = 0; j < WARPS; ++j) {
                heats_local[i][j] = 0.0f;
                heats_squared_local[i][j] = 0.0f;
	    }
        }
    }
    __syncthreads();

    // Fase 2: Cómputo
    photon(heats_local[wtid], heats_squared_local[wtid], &rng);
    __syncthreads();


    // Fase 3.1: Acumulación por warp
    if (threads_this_frame > warpSize)
    for (int offset = WARPS / 2; offset > 0; offset /= 2) {
        if (wtid < offset) {
            for (int i = 0; i < SHELLS; ++i) {
                heats_local[wtid][i] += heats_local[wtid + offset][i];
                heats_squared_local[wtid][i] += heats_squared_local[wtid + offset][i];
            }
        }
        __syncthreads();
    }

    // Fase 3.2: Acumulación por bloque
    if (btid == 0) {
	for (int i = 0; i < SHELLS; ++i) {    
            atomicAdd(&heats[i], heats_local[0][i]);
            atomicAdd(&heats_squared[i], heats_squared_local[0][i]);
	}
    }
}


int main(void) {
    assert(THREADS_PER_BLOCK >= SHELLS && "THREADS_PER_BLOCK debe ser mayor o igual a SHELLS");

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_VISIBLE, GLFW_FALSE);
    GLFWwindow* window = glfwCreateWindow(800, 800, "tiny mc", NULL, NULL);
    assert(window);
    glfwMakeContextCurrent(window);
   
    if (glewInit() != GLEW_OK) {
        fprintf(stderr, "GLEW initialization failed\n");
        return -1;
    }


    GLuint vshader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vshader, 1, &VSHADER, NULL);
    glCompileShader(vshader);
    GL_CHECK();
    GLint status;
    glGetShaderiv(vshader, GL_COMPILE_STATUS, &status);
    assert(status);

    GLuint fshader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fshader, 1, &FSHADER, NULL);
    glCompileShader(fshader);
    GL_CHECK();
    glGetShaderiv(fshader, GL_COMPILE_STATUS, &status);
    assert(status);

    GLuint program = glCreateProgram();
    glAttachShader(program, vshader);
    glAttachShader(program, fshader);
    glLinkProgram(program);
    GL_CHECK();
    glGetProgramiv(program, GL_LINK_STATUS, &status);
    assert(status);

    glDeleteShader(vshader);
    glDeleteShader(fshader);
    glUseProgram(program);

    GLuint vao;
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);
    glViewport(0, 0, 800, 800);

    GLuint ssbo_heats, ssbo_heats_squared;
    hipGraphicsResource* cuda_res_heats;
    hipGraphicsResource* cuda_res_heats_squared;

    glGenBuffers(1, &ssbo_heats);
    glBindBuffer(GL_SHADER_STORAGE_BUFFER, ssbo_heats);
    glBufferData(GL_SHADER_STORAGE_BUFFER, SHELLS * sizeof(float), NULL, GL_DYNAMIC_DRAW);
    glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 0, ssbo_heats);
    glFinish();

    glGenBuffers(1, &ssbo_heats_squared);
    glBindBuffer(GL_SHADER_STORAGE_BUFFER, ssbo_heats_squared);
    glBufferData(GL_SHADER_STORAGE_BUFFER, SHELLS * sizeof(float), NULL, GL_DYNAMIC_DRAW);
    glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 1, ssbo_heats_squared);
    glFinish();

    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cuda_res_heats, ssbo_heats, cudaGraphicsMapFlagsWriteDiscard));
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cuda_res_heats_squared, ssbo_heats_squared, cudaGraphicsMapFlagsWriteDiscard));

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(simulate_kernel), hipFuncCachePreferL1); 

    glfwShowWindow(window);

    unsigned long remaining_photons = PHOTON_CAP;
    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();

        if (remaining_photons > 0) {
	    unsigned long photons_this_frame = PHOTONS_PER_THREAD * THREADS_PER_FRAME;
	    if(photons_this_frame > remaining_photons) photons_this_frame = remaining_photons;
	    remaining_photons -= photons_this_frame;
		
	    glFinish();

            hipGraphicsResource* resources[] = { cuda_res_heats, cuda_res_heats_squared };
	    CUDA_CHECK(hipGraphicsMapResources(2, resources));

	    float* dev_heats;
	    float* dev_heats_squared;
	    size_t size;

	    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&dev_heats, &size, cuda_res_heats));
            CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&dev_heats_squared, &size, cuda_res_heats_squared));

	    int threads = THREADS_PER_BLOCK;
	    int blocks = (photons_this_frame + threads - 1) / threads;
	    simulate_kernel<<<blocks, threads>>>(dev_heats, dev_heats_squared, THREADS_PER_FRAME);
	    hipDeviceSynchronize();

	    CUDA_CHECK(hipGraphicsUnmapResources(2, resources));
        }

        glClear(GL_COLOR_BUFFER_BIT);
        glDrawArrays(GL_TRIANGLES, 0, 6);
        glfwSwapBuffers(window);
    }

    CUDA_CHECK(hipGraphicsUnregisterResource(cuda_res_heats));
    CUDA_CHECK(hipGraphicsUnregisterResource(cuda_res_heats_squared));

    glDeleteBuffers(1, &ssbo_heats);
    glDeleteBuffers(1, &ssbo_heats_squared);
    glDeleteVertexArrays(1, &vao);
    glDeleteProgram(program);
    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
