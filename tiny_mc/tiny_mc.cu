#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#include "params.h"
#include "photon.cuh"
#include "xorshift32.cuh"


__global__ void simulate_kernel(float* __restrict__ heats, float* __restrict__ heats_squared)
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gtid >= PHOTONS) return;
    int btid = threadIdx.x;

    // Fase 1: Incialización
    __shared__ float heats_local[SHELLS];
    __shared__ float heats_squared_local[SHELLS];
    Xorshift32 rng;
    xorshift32_init(&rng, SEED ^ (btid * 0x9E3779B9u) ^ (blockIdx.x * 0x85EBCA6Bu) ^ (blockDim.x * 0xC2B2AE35u));
    if (btid < SHELLS) { // OJO! Solo funciona si THREADS_PER_BLOCK >= SHELLS
        heats_local[btid] = 0.0f;
        heats_squared_local[btid] = 0.0f;
    }
    __syncthreads();

    // Fase 2: Cómputo
    photon(heats_local, heats_squared_local, &rng);
    __syncthreads();

    // Fase 3: Acumulación
    if (btid < SHELLS) { // OJO! Solo funciona si THREADS_PER_BLOCK >= SHELLS
        atomicAdd(&heats[btid], heats_local[btid]);
        atomicAdd(&heats_squared[btid], heats_squared_local[btid]);
    }
}


int main() { 
    
    assert(THREADS_PER_BLOCK >= SHELLS && "THREADS_PER_BLOCK debe ser mayor o igual a SHELLS");

    const int size = SHELLS * sizeof(float);

    float* d_heats;
    float* d_heats_squared;
    hipMalloc(&d_heats, size);
    hipMalloc(&d_heats_squared, size);
    hipMemset(d_heats, 0, size);
    hipMemset(d_heats_squared, 0, size);
    
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);

    // Reservamos toda la RAM libre del GPU para que nadie mas la usea MUAJAJAJAJ
    //void* gobble;
    //hipMalloc(&gobble, free_mem - (16 * 1024 * 1024)); // dejamos 16 MiB libres por seguridad

    float elapsed_time;
    hipEvent_t e1, e2;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventRecord(e1);

    simulate_kernel<<<(PHOTONS/THREADS_PER_BLOCK)/PHOTONS_PER_THREAD, THREADS_PER_BLOCK>>>(d_heats, d_heats_squared);
    hipDeviceSynchronize();

    hipEventRecord(e2);
    hipEventSynchronize(e2);
    hipEventElapsedTime(&elapsed_time, e1, e2);

    float* heats = (float*)malloc(size);
    hipMemcpy(heats, d_heats, size, hipMemcpyDeviceToHost);
    float* heats_squared = (float*)malloc(size);
    hipMemcpy(heats_squared, d_heats_squared, size, hipMemcpyDeviceToHost);

    printf("%f\n", PHOTONS / (1000.0f * elapsed_time));

    // printf("# Radius\tHeat\n");
    // printf("# [microns]\t[W/cm^3]\tError\n");
    // float t = 4.0f * M_PI * powf(MICRONS_PER_SHELL, 3.0f) * PHOTONS / 1e12;
    // for (unsigned int i = 0; i < SHELLS - 1; ++i) {
    //     printf("%6.0f\t%12.5f\t%12.5f\n", i * (float)MICRONS_PER_SHELL,
    //            heats[i] / t / (i * i + i + 1.0 / 3.0),
    //            sqrt(heats_squared[i] - heats[i] * heats[i] / PHOTONS) / t / (i * i + i + 1.0f / 3.0f));
    // }
    // printf("# extra\t%12.5f\n", heats[SHELLS - 1] / PHOTONS);

    hipFree(d_heats);
    hipFree(d_heats_squared);
    free(heats);
    free(heats_squared);

    return 0;
}
