#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "params.h"
#include "photon.cuh"
#include "xorshift32.cuh"


__global__ void simulate_kernel(float* heats, float* heats_squared, unsigned int photons, unsigned int seed)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= photons) return;

    Xorshift32 rng;
    xorshift32_init(&rng, seed ^ tid);  // semilla única por hilo

    photon(heats, heats_squared, &rng);
}

int main()
{
    const int size = SHELLS * sizeof(float);

    float* d_heats;
    float* d_heats_squared;
    hipMalloc(&d_heats, size);
    hipMalloc(&d_heats_squared, size);
    hipMemset(d_heats, 0, size);
    hipMemset(d_heats_squared, 0, size);

    double elapsed_time;
    hipEvent_t e1, e2;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventRecord(e1);

    simulate_kernel<<<PHOTONS/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_heats, d_heats_squared, num_photons, SEED);
    hipDeviceSynchronize();

    hipEventRecord(e2);
    hipEventSynchronize(e2);
    hipEventElapsedTime(&elapsed_time, e1, e2);

    float* heats = (float*)malloc(size);
    hipMemcpy(heats, d_heats, size, hipMemcpyDeviceToHost);

    printf("%f\n", 1e-3 * PHOTONS / elapsed_time); // Kfotones / ms

    hipFree(d_heats);
    hipFree(d_heats_squared);
    free(heats);

    return 0;
}
