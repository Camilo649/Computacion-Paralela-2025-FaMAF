#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "params.h"
#include "photon.cuh"
#include "xorshift32.cuh"


__global__ void simulate_kernel(float* heats, float* heats_squared)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= PHOTONS) return;
    Xorshift32 rng;
    xorshift32_init(&rng, (SEED ^ tid) + 1);  // semilla única por hilo
    photon(heats, heats_squared, &rng);
}


int main()
{
    const int size = SHELLS * sizeof(float);

    float* d_heats;
    float* d_heats_squared;
    hipMalloc(&d_heats, size);
    hipMalloc(&d_heats_squared, size);
    hipMemset(d_heats, 0, size);
    hipMemset(d_heats_squared, 0, size);

    float elapsed_time;
    hipEvent_t e1, e2;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventRecord(e1);

    simulate_kernel<<<PHOTONS/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_heats, d_heats_squared);
    hipDeviceSynchronize();

    hipEventRecord(e2);
    hipEventSynchronize(e2);
    hipEventElapsedTime(&elapsed_time, e1, e2);

    float* heats = (float*)malloc(size);
    hipMemcpy(heats, d_heats, size, hipMemcpyDeviceToHost);
    float* heats_squared = (float*)malloc(size);
    hipMemcpy(heats_squared, d_heats_squared, size, hipMemcpyDeviceToHost);

    printf("%f\n", PHOTONS / (1000.0f * elapsed_time));

    hipFree(d_heats);
    hipFree(d_heats_squared);
    free(heats);
    free(heats_squared);

    return 0;
}
