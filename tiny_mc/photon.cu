#include "hip/hip_runtime.h"
#pragma once

#include <math.h>
#include <hip/hip_runtime.h>

#include "params.h"
#include "xorshift32.cuh"


__device__ void photon(float* heats, float* heats_squared, Xorshift32* rng)
{
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4 / MICRONS_PER_SHELL / (MU_A + MU_S);

    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    for (;;) {
        float t = -logf(xorshift32_norm(rng));
        x += t * u;
        y += t * v;
        z += t * w;

        unsigned int shell = sqrtf(x * x + y * y + z * z) * shells_per_mfp;
        if (shell >= SHELLS) shell = SHELLS - 1;

        float absorb = (1.0f - albedo) * weight;
        atomicAdd(&heats[shell], absorb);
        atomicAdd(&heats_squared[shell], absorb * absorb);

        weight *= albedo;

        float xi1, xi2;
        do {
            xi1 = 2.0f * xorshift32_norm(rng) - 1.0f;
            xi2 = 2.0f * xorshift32_norm(rng) - 1.0f;
            t = xi1 * xi1 + xi2 * xi2;
        } while (t > 1.0f);
        u = 2.0f * t - 1.0f;
        float sqrt_term = sqrtf((1.0f - u * u) / t);
        v = xi1 * sqrt_term;
        w = xi2 * sqrt_term;

        if (weight < 0.001f) {
            if (xorshift32_norm(rng) > 0.1f) break;
            weight /= 0.1f;
        }
    }
}
